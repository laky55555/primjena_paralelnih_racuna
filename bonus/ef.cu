#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <sstream>
#include "CImg.h"
#include  "cuda_wrapper.h"
#include "hipblas.h"
#include "magma.h"
#include "magma_lapack.h"

using namespace cimg_library;


/**
 * Auxiliary function for printing first few rows of matrix.
 * @param matrix Matrix we want to print.
 * @param m      Number of rows in matrix.
 * @param n      Number of columns in matrix.
 */
void print_matrix(float *matrix, int m, int n)
{
	for(int i = 0; i < 5; i++)
	{
		for(int j = 0; j < 10; j++)
			printf("%f ", matrix[i*n + j]);
		printf("\n");
	}
}

/**
 * Function for converting number to string.
 * @param  number Number to convert to string.
 * @return String Input number in string form.
 */
std::string number_to_string(int number)
{
	std::ostringstream convert;
	convert << number;
	return convert.str();
}


/**
 * Auxiliary function for loading picture data from memory to GPU memory.
 * Function also calculate mean value for each pixel of pictures and save
 * it on device on dev_mean
 * @param  directory  Parent directory in which we can find all pictures.
 * @param  pic_number Number of pictures we will load into memory
 * @param  pic_dim    Dimension of each picture. All pictures have the same resolution.
 * @param  hst_matrix Data from all pictures saved in host memory.
 * @param  dev_matrix Data from all pictures saved in device memory.
 * @param  dev_mean      Mean value for each pixel saved in device vector.
 * @param  handle     initialized handle for cublas operations.
 * @param  hst_target Data from picture we are looking for, copying from host to device.
 * @param  dev_target Data from picture we are looking for.
 * @return int        Returns 0 if everything is OK.
 */
int load_pictures(const char *directory, int pic_number, int pic_dim, float *hst_matrix, float *dev_matrix, float *dev_mean, hipblasHandle_t handle, float *hst_target, float *dev_target)
{

	float alfa = 1./pic_number;

	for(int i = 0; i < pic_number; i++)
	{
		std::string pic_name(std::string (directory) + "/s" + number_to_string(i/10+1) + "/" + number_to_string(i%10+1) + ".pgm");
		CImg<float> picture(pic_name.c_str());
		for(int j = 0; j < pic_dim; j++)
			hst_matrix[i*pic_dim + j] = picture.data()[j];

		//Copying data onto device and accumulating vector sum in one vector; maybe it can be done in parallel.
		cuda_exec(hipMemcpy(i*pic_dim + dev_matrix, i*pic_dim + hst_matrix, pic_dim * sizeof(float), hipMemcpyHostToDevice));

		//Accumulating mean over each pixel in vector dev_mean.
		cublas_exec(hipblasSaxpy(handle, pic_dim, &alfa, i*pic_dim + dev_matrix, 1, dev_mean, 1 ));

	}

	cuda_exec(hipMemcpy(dev_target, hst_target, pic_dim * sizeof(float), hipMemcpyHostToDevice));

	return 0;
}

/**
 * Function for calculating singular value decomposition (SVD).
 * Function gets and return all values through arguments.
 * 					M = U * S * V^T
 * http://icl.cs.utk.edu/projectsfiles/magma/doxygen/group__magma__gesvd.html#ga96e26a734d9c48e5c994863c4e2d83f1
 * @param  pic_dim     Number of rows in matrix for SVD.
 * @param  pic_number  Number of cols in matrix for SVD.
 * @param  matrix      Matrix for SVD.
 * @param  sing_values Rectangular diagonal matrix with non-negative real numbers on the diagonal.
 *                     The singular values of matrix, sorted so that S(i) >= S(i+1)
 * @param  U		   Unitary matrix.
 * @param  V           Unitary matrix.
 * @return int         Returns 0 if everything is OK.
 */
int calculating_svd(int pic_dim, int pic_number, float *matrix, float *sing_values, float *U, float *V)
{
	magma_init();

	int info;
	int lwork = -1;
	float *work;
	host_alloc(work, float, sizeof(float));

	//
	magma_sgesvd(MagmaAllVec, MagmaAllVec, pic_dim, pic_number, matrix, pic_dim, sing_values, U, pic_dim, V, pic_number, work, lwork, &info);
	lwork = work[0];
	host_alloc(work, float, lwork * sizeof(float));
	magma_sgesvd(MagmaAllVec, MagmaAllVec, pic_dim, pic_number, matrix, pic_dim, sing_values, U, pic_dim, V, pic_number, work, lwork, &info);

	magma_finalize();

	return 0;
}


/**
 * Program need 2 parameters, first is location of picture you want to find, and
 * second is directory location for training set from which we will find the closest
 * looking picture for one given in first parameter.
 * All input pictures need to be the same size and resolution. All pictures need
 * to be black and white.
 * @param  argc Number of parameters.
 * @param  argv Array of strings which hold parameters.
 * @return int 	Returns 0 if everything is OK.
 */
int main(int argc, char const *argv[])
{

	//Program needs 3 parameters for functioning correctly.
	if(argc != 3)
	{
		fprintf(stderr, "Program %s expects: picture location, directory with test pictures.\n", argv[0]);
		return -1;
	}

	//Loading target picture for checking pictures dimensions.
	CImg<float> target(argv[1]);

	//Number of pictures for training set.
	unsigned int pic_number = 400;
	unsigned int pic_width = target.width();
	unsigned int pic_height = target.height();
	unsigned int pic_dim = pic_height * pic_width;


	//Defining all needed pointers.
	float *hst_matrix;
	float *hst_norms;

	float *hst_sing_values;
	float *hst_u;
	float *hst_v;

	float *dev_target;
	float *dev_mean;
	float *dev_matrix;
	float *dev_u;
	float *dev_result_matrix;
	float *dev_result_target;
	float *dev_norms;

	hipblasHandle_t handle;

	//Allocating host memory.
	host_alloc(hst_matrix, float, pic_dim * pic_number * sizeof(float));
	host_alloc(hst_norms, float, pic_number * sizeof(float));

	host_alloc(hst_sing_values, float, pic_number * sizeof(float));
	host_alloc(hst_u, float, pic_dim * pic_dim * sizeof(float));
	host_alloc(hst_v, float, pic_number * pic_number * sizeof(float));

	//Allocating device memory.
	cuda_exec(hipMalloc(&dev_target, pic_dim * sizeof(float)));
	cuda_exec(hipMalloc(&dev_mean, pic_dim * sizeof(float)));
	cuda_exec(hipMalloc(&dev_matrix, pic_dim * pic_number * sizeof(float)));
	cuda_exec(hipMalloc(&dev_u, pic_dim * pic_dim * sizeof(float)));
	cuda_exec(hipMalloc(&dev_result_matrix, pic_dim * pic_number * sizeof(float)));
	cuda_exec(hipMalloc(&dev_result_target, pic_dim * sizeof(float)));
	cuda_exec(hipMalloc(&dev_norms, pic_number * sizeof(float)));

	cuda_exec(hipMemset(dev_mean, 0, pic_dim * sizeof(float)));

	//Initializing of handle for cublas.
	cublas_exec(hipblasCreate(&handle));
	cublas_exec(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));


	//Loading pictures in device memory and calculating mean value per pixel.
	load_pictures(argv[2], pic_number, pic_dim, hst_matrix, dev_matrix, dev_mean, handle, target.data(), dev_target);

	//Normalization of pictures data values.
	//	matrix = matrix - mean value per each element.
	//	vector = vector - mean value per each element.
	float alfa = -1;
	for(int i = 0; i < pic_number; i++)
		cublas_exec(hipblasSaxpy(handle, pic_dim, &alfa, dev_mean, 1, i*pic_dim + dev_matrix, 1 ));

	cublas_exec(hipblasSaxpy(handle, pic_dim, &alfa, dev_mean, 1, dev_target, 1 ));


	//Returning of newly calculated matrix in host memory (needed for magma svd calculating).
	cuda_exec(hipMemcpy(hst_matrix, dev_matrix, pic_dim * pic_number * sizeof(float), hipMemcpyDeviceToHost));

	//Calculating unitary matrix U from SVD.
	calculating_svd(pic_dim, pic_number, hst_matrix, hst_sing_values, hst_u, hst_v);
	//Copying matrix U from host to device because magma returns values on host..
	cuda_exec(hipMemcpy(dev_u, hst_u, pic_dim * pic_dim * sizeof(float), hipMemcpyHostToDevice));


	//y = U^T * matrix
	//Eigenface representation for mean matrix and target picture.
	alfa = 1;
	float beta = 0;
	cublas_exec(hipblasSgemm(handle,  HIPBLAS_OP_T,  HIPBLAS_OP_N, pic_dim, pic_number, pic_dim, &alfa, dev_u, pic_dim, dev_matrix, pic_dim, &beta, dev_result_matrix, pic_dim));
	cublas_exec(hipblasSgemm(handle,  HIPBLAS_OP_T,  HIPBLAS_OP_N, pic_dim, 1, pic_dim, &alfa, dev_u, pic_dim, dev_target, pic_dim, &beta, dev_result_target, pic_dim));


	//Subtraction target picture from training set data for looking the closest one.
	//Finding closest one by looking for norm2 on each row.
	//Dividing each picture from training set with our target picture so we can find
	//the most similar one.
	//Each row norm is saved on host memory (better on device if it's possible).
	alfa = -1;
	float res;
	//Maybe could go in parallel.
	for(int i = 0; i < pic_number; i++) {
		cublas_exec(hipblasSaxpy(handle, pic_dim, &alfa, dev_result_target, 1, i*pic_dim + dev_result_matrix, 1 ));
        cublas_exec(hipblasSnrm2(handle, pic_dim, i*pic_dim + dev_result_matrix, 1, hst_norms+i));
	}

	//Searching for exact column that is closest by norm2 to our target picture.
	int result;
	cuda_exec(hipMemcpy(dev_norms, hst_norms, pic_number * sizeof(float), hipMemcpyHostToDevice));
	hipblasIsamin(handle, pic_number, dev_norms, 1, &result);
	result--;

	printf("Target picture is in directory s%d, and closest match is picture %d.pgm\n", result/10+1, result%10+1);

//Deallocating memory
end:
	free(hst_matrix);
	free(hst_norms);

	free(hst_sing_values);
	free(hst_u);
	free(hst_v);

	cuda_exec(hipFree(dev_target));
	cuda_exec(hipFree(dev_mean));
	cuda_exec(hipFree(dev_matrix));
	cuda_exec(hipFree(dev_u));
	cuda_exec(hipFree(dev_result_matrix));
	cuda_exec(hipFree(dev_result_target));
	cuda_exec(hipFree(dev_norms));

	cublas_exec(hipblasDestroy(handle));

	return 0;
}
